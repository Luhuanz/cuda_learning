#include "hip/hip_runtime.h"
#include<iostream>

__global__ void vectorAdd(int *a,int* b,int *c,int len_vector){
  int i=threadIdx.x;
  if(i<len_vector){
    c[j]=a[i]+b[i];
  }
}


int main(){

  int len_vector=32
  int vector_1[len_vector],vector_2[len_vector],vector_3[len_vector];
  int *vector_1_gpu,*vector_2_gpu,*vector_3_gpu;  
  // 赋值
  for(int i=0;i<len_vector;i++){
    vector_1[i]=1;
    vector_2[i]=2;
  }
  // 打印 两个向量
  printf("vector 1:\n");
  for(int j=0;j<len_vector;++j){
    printf("%d",vector_1[j]);
  }
  printf("vector2:\n");
  for(int j=0;j<len_vector;++j){
    printf("%d",vector2[j]);
  }
  hipMalloc((void**)&vector_1_gpu,len_vector*sizeof(int));
  hipMalloc((void**)&vector_2_gpu,len_vector*sizeof(int));
  hipMalloc((void**)&vector_3_gpu,len_vector*sizeof(int));
  //host to device
  hipMemcpy(vector_1_gpu,vector_1,len_vector*sizeof(int),hipMemcpyHostToDevice)
  hipMemcpy(vector_2_gpu,vector_2,len_vector*sizeof(int),hipMemcpyHostToDevice)
  hipMemcpy(vector_3_gpu,vector_3,len_vector*sizeof(int),hipMemcpyHostToDevice)

// run kernel
int threadNum=len_vector; // 设置核函数的thread


vectorAdd(vector_1_gpu,vector_2_gpu,vector_3_gpu,len_vector);




}
